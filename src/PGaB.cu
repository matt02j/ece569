///////////////////////////////////////////////////////////////////////////////////////////////////
// Created By        : Adithya Madhava Rao, Harshil Pankaj Kakaiya, Priyanka Devashish Goswami
// Modified By       : Matt <LastName>, Jeremy Seers, Sebastian Thiem
//                   :
// Organization:     : The University of Arizona
//                   :
// Project Name:     : OPTIMIZATIONS OF LDPC DECODERS IN CUDA
// File Name:        : PGaB.cu
//                   :
// Create Date:      : 8 May 2017
// Modified          : 24 March 2019
//                   :
// Description:      : Probabalistic Gallager B
//                   :
///////////////////////////////////////////////////////////////////////////////////////////////////

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 

#include <unistd.h> 
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
// #include <omp.h>

#define arrondi(x)((ceil(x) - x) < (x - floor(x)) ? (int) ceil(x) : (int) floor(x))
#define min(x, y)((x) < (y) ? (x) : (y))
#define signf(x)((x) >= 0 ? 0 : 1)
#define max(x, y)((x) < (y) ? (y) : (x))
#define SQR(A)((A) * (A))
#define BPSK(x)(1 - 2 * (x))
#define PI 3.1415926536
   
#define PROFILE

__constant__ int Mat_device[5184];

// TODO Verify stride paterns
// All of the following functions use this
// for (unsigned stride = 0; stride < (NbBranch / N); stride++)
// pattern and this seems inherently a little weird that each
// thread is striding 1, 2, 3, 4... away from itself...
// 
// TODO The Data pass modules and their 2 for loops essentially 
// read CtoV[node_idx] across "strides" different elements, and 
// throws away data so the next for loop has to read global again. 
// Maybe find a way to keep these values in shared memory. This may 
// be weird because of the striding pattern, but if that can be 
// addressed this opens up for us
//
// TODO once we model what these functions are doing 
// COMMENT
// 
// TODO GaussianElimination_MRB as a kernel
//
// TODO Main is reading args in the most horendous way ive
// ever seen, us fstreams, arg parser, ect anything to make
// that not what it is
// also uses 4 different input files, 3 of which are only dimensions
//
// TODO replace calloc with mallocs where possible
//
// also uses 4 different input files, 3 of which are only dimensions
// TODO replace calloc with mallocs where possible
//
// TODO make sure all allocated memory gets freed at the apropriate time

// Message from channel copied into variable node to check node array.
__global__ void DataPassGB_0(int * VtoC, int * Receivedword, int * Interleaver, int N, int NbBranch) {
   
   // calculate the current index on the grid
   unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

   if (id < N) {

      // 
      unsigned node_idx = 0;

      // 
      unsigned strides = (NbBranch / N);

      // 
      unsigned i = Receivedword[id];

      for (unsigned stride = 0; stride < strides; stride++) {
         
         // get node index from interleaver
         node_idx = Interleaver[id * strides + stride];
         
         VtoC[node_idx] = i;
      }
   }
}

// for iterations between 1 and 15, this kernel launches to pass the message from variables nodes onto 
// the four check nodes it is connected to.
__global__ void DataPassGB_1(int* VtoC, int* CtoV, int* Receivedword, int* Interleaver, int N, int NbBranch) {
   
   // calculate the current index on the grid
   unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

   if (id < N) {

      // 
      int buf = 0;

      // 
      int i = Receivedword[id];

      // 
      int Global = (1 - 2 * i);

      // Used to index the CtoV and VtoC node arrays
      unsigned node_idx = 0;

      // 
      unsigned strides = (NbBranch / N);

      // 
      for (unsigned stride = 0; stride < strides; stride++) {

         // get node index from interleaver
         node_idx = Interleaver[id * strides + stride];

         // 
         Global += (-2) * CtoV[node_idx] + 1;
      }

      // 
      for (unsigned stride = 0; stride < strides; stride++) {

         // get node index from interleaver
         node_idx = Interleaver[id * strides + stride];

         // 
         // 
         buf = Global - ((-2) * CtoV[node_idx] + 1);
         
         // 
         VtoC[node_idx] = (buf < 0)? 1 : ((buf > 0)? 0 : i);
      }
   }
}

// for iterations greater than 15, this kernel launches to pass the message from variables nodes onto the four 
// check nodes it is connected to.
__global__ void DataPassGB_2(int* VtoC, int* CtoV, int* Receivedword, int* Interleaver, int N, int NbBranch, int varr) {
   
   // calculate the current index on the grid
   unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

   if (id < N) {

      // 
      int buf;
      
      // 
      int i = Receivedword[id];

      // 
      int Global = (1 - 2 * (varr ^ i));

      // 
      unsigned node_idx = 0;

      //
      unsigned strides = (NbBranch / N);

      //
      for (unsigned stride = 0; stride < strides; stride++) {

         // calculate node index
         node_idx = Interleaver[id * strides + stride];

         Global += (-2) * CtoV[node_idx] + 1;
      }

      // 
      for (unsigned stride = 0; stride < strides; stride++) {
         
         // calculate node index
         node_idx = Interleaver[id * strides + stride];

         // 
         // 
         buf = Global - ((-2) * CtoV[node_idx] + 1);

         // 
         VtoC[node_idx] = (buf < 0)? 1 : ((buf > 0)? 0 : i);
      }
   }
}

// This kernel is launched to check if the CtoV copies the same information as VtoC depending upon the signe value
__global__ void CheckPassGB(int* CtoV, int* VtoC, int M, int NbBranch) {
  
   // calculate the current index on the grid
   unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

   if (id < M) {

      int signe = 0;

      // For indexing the node arrays
      unsigned node_idx = 0;

      // 
      unsigned strides = (NbBranch / M);
      
      // 
      for (unsigned stride = 0; stride < strides; stride++) {

         node_idx = stride + id * strides;
         signe ^= VtoC[node_idx];
      }
      
      // 
      for (unsigned stride = 0; stride < strides; stride++) {
         
         node_idx = stride + id * strides;
         CtoV[node_idx] = signe ^ VtoC[node_idx];
      }
   }
}

// The following kernel is launched to decide each check node's decision whether the corresponding variable nodes 
// are in error or not.
__global__ void APP_GB(int* Decide, int* CtoV, int* Receivedword, int* Interleaver, int N, int NbBranch) {
   
   // calculate the current index on the grid
   unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

   if (id < N) {

      // 
      int i = Receivedword[id];

      // 
      int Global = (1 - 2 * i);

      // Used to index the node array
      unsigned node_idx = 0;

      // 
      unsigned strides = (NbBranch / N);

      // 
      for (unsigned stride = 0; stride < strides; stride++) {

         // TODO this is not coalesced at all
         node_idx = Interleaver[id * strides + stride];
         Global += (-2) * CtoV[node_idx] + 1;
      }
      
      // 
      Decide[id] = (Global < 0)? 1 : ((Global > 0)? 0 : i);
   }
}

//Here a cumulative decision is made on the variable node error depending upon all the four check nodes to which the variable node is connected to 
__global__ void ComputeSyndrome(int * Synd, int * Decide, int M, int NbBranch) {

   // calculate the current index on the grid
   unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

   // intialize ___ regardless of bounds...
   int synd = 0;
   
   if (id < M) {
      
      unsigned strides = (NbBranch / M);

      // 
      for (unsigned stride = 0; stride < strides; stride++) {

         __syncthreads();

         synd ^=Decide[Mat_device[id * strides + stride]];
      }
   }

   // NOTE write back regardless of thread
   Synd[id]=synd;
}

// 
unsigned GaussianElimination_MRB(int* Perm, int** MatOut, int** Mat, int M, int N) {
   
   // 
   int buf;
   
   // 
   // used in a for loop with break
   // TODO bad practice, fix for loop if possible
   unsigned ind;

   // 
   unsigned Rank;

   // 
   int* Index;
   Index = (int*) calloc(N, sizeof(int));

   // Triangularization
   int indColumn = 0;
   int nb = 0;
   int dep = 0;

   //
   for (unsigned m = 0; m < M; m++) {

      // 
      if (indColumn == N) {
         dep = M - m;
         break;
      }

      // 
      for (ind = m; ind < M; ind++) {
         if (Mat[ind][indColumn] != 0) {
            break;
         }
      }

      // If a "1" is found on the column, permutation of rows
      if (ind < M) {

         // swap row "m" with row "ind" from "indColumn" to the end of the row
         for (unsigned n = indColumn; n < N; n++) {
            buf = Mat[m][n];
            Mat[m][n] = Mat[ind][n];
            Mat[ind][n] = buf;
         }

         // bottom of the column ==> 0
         for (unsigned m1 = m + 1; m1 < M; m1++) {

            // 
            if (Mat[m1][indColumn] == 1) {

               // XOR row "m1" with row "m" from "indColumn" to the end of the row
               for (unsigned n = indColumn; n < N; n++) {
                  Mat[m1][n] = Mat[m1][n] ^ Mat[m][n];
               }
            }
         }

         Perm[m] = indColumn;
      }
      else { 

         // else we "mark" the column.
         Index[nb++] = indColumn;
         m--;
      }

      indColumn++;
   }

   // 
   Rank = M - dep;

   // 
   for (unsigned n = 0; n < nb; n++) {
      Perm[Rank + n] = Index[n];
   }

   // Permutation of the matrix
   for (unsigned m = 0; m < M; m++) {
      for (unsigned n = 0; n < N; n++) {
         MatOut[m][n] = Mat[m][Perm[n]];
      }
   }

   // Diagonalization
   for (unsigned m = 0; m < (Rank - 1); m++) {
      for (unsigned n = m + 1; n < Rank; n++) {
         
         //
         if (MatOut[m][n] == 1) {
            for (unsigned k = n; k < N; k++) {
               MatOut[m][k] = MatOut[n][k] ^ MatOut[m][k];
            }
         }
      }
   }

   free(Index);

   return Rank;
}

unsigned long diff_time_usec(struct timeval start, struct timeval stop){
  unsigned long diffTime;
  if(stop.tv_usec < start.tv_usec){
   diffTime = 1000000 + stop.tv_usec-start.tv_usec;
        diffTime += 1000000 * (stop.tv_sec - 1 - start.tv_sec);
  }
  else{
   diffTime = stop.tv_usec - start.tv_usec;
        diffTime += 1000000 * (stop.tv_sec - start.tv_sec);
  }
  return diffTime;
}

int main(int argc, char * argv[]) {
   if(argc < 3 ){
      fprintf(stderr,"Usage: PGaB /Path/To/Data/File Path/to/output/file");
   }

   struct timeval start,stop;
   unsigned long diffTime=0;

   // 
   FILE * f;

   // 
   int Graine;

   unsigned NbIter;

   unsigned nbtestedframes;

   unsigned NBframes;

   // 
   float alpha_max, alpha_min, alpha_step, alpha, NbMonteCarlo;

   // ----------------------------------------------------
   // read command line params
   // ----------------------------------------------------
   char* FileName;
   char* FileMatrix;
   char* FileResult;
   FileName = (char * ) malloc(200);
   FileMatrix = (char * ) malloc(200);
   FileResult = (char * ) malloc(200);

   strcpy(FileMatrix, argv[1]); // Matrix file
   strcpy(FileResult, argv[2]); // Results file
   //--------------Simulation input for GaB BF-------------------------
   NbMonteCarlo = 1000000; // Maximum nb of codewords sent
   NbIter = 100; // Maximum nb of iterations
   alpha = 0.01; // Channel probability of error
   NBframes = 100; // Simulation stops when NBframes in error
   Graine = 1; // Seed Initialization for Multiple Simulations

   // shortend for testing purposes, was alpha_max=0.06
   alpha_max = 0.03; //Channel Crossover Probability Max and Min
   alpha_min = 0.03;
   alpha_step = 0.01;

   // ----------------------------------------------------
   // Load Matrix
   // ----------------------------------------------------
   int * ColumnDegree, * RowDegree, ** Mat_host, * Mat_host1;
   int M, N, m, n, k;
   strcpy(FileName, FileMatrix);
   strcat(FileName, "_size");
   f = fopen(FileName, "r");
   fscanf(f, "%d", & M);
   fscanf(f, "%d", & N);
   ColumnDegree = (int * ) calloc(N, sizeof(int));
   RowDegree = (int * ) calloc(M, sizeof(int));
   fclose(f);
   strcpy(FileName, FileMatrix);
   strcat(FileName, "_RowDegree");
   f = fopen(FileName, "r");

   for (m = 0; m < M; m++) {
      fscanf(f, "%d", & RowDegree[m]);
   }
   fclose(f);

   Mat_host = (int ** ) calloc(M, sizeof(int * ));

   for (m = 0; m < M; m++) {
      Mat_host[m] = (int * ) calloc(RowDegree[m], sizeof(int));
   }

   //changes made
   strcpy(FileName, FileMatrix);

   f = fopen(FileName, "r");
   for (m = 0; m < M; m++) {
      for (k = 0; k < RowDegree[m]; k++) {
         fscanf(f, "%d", & Mat_host[m][k]);
      }
   }
   fclose(f);

   for (m = 0; m < M; m++) {
      for (k = 0; k < RowDegree[m]; k++){
         ColumnDegree[Mat_host[m][k]]++;
      }
   }
   //TODO free filename and filematrix
   printf("Matrix Loaded \n");

   // ----------------------------------------------------
   // Build Graph
   // ----------------------------------------------------
   int NbBranch, ** NtoB, * Interleaver_host, * ind, numColumn, numBranch;
   
   NbBranch = 0;
   
   for (m = 0; m < M; m++) {
      NbBranch = NbBranch + RowDegree[m];
   }
   
   NtoB = (int ** ) calloc(N, sizeof(int * ));
   
   for (n = 0; n < N; n++) {
      NtoB[n] = (int * ) calloc(ColumnDegree[n], sizeof(int));
   }
   
   Interleaver_host = (int * ) calloc(NbBranch, sizeof(int));
   ind = (int * ) calloc(N, sizeof(int));
   numBranch = 0;
   
   for (m = 0; m < M; m++) {
      for (k = 0; k < RowDegree[m]; k++) {
         numColumn = Mat_host[m][k];
         NtoB[numColumn][ind[numColumn]++] = numBranch++;
      }
   }

   free(ind);
   numBranch = 0;

   for (n = 0; n < N; n++) {
      for (k = 0; k < ColumnDegree[n]; k++) {
         Interleaver_host[numBranch++] = NtoB[n][k];
      }
   }
   
   Mat_host1 = (int * ) calloc(NbBranch, sizeof(int));
   
   for (m = 0; m < M; m++) {
      for (n = 0; n < 8; n++) {
         Mat_host1[m * 8 + n] = Mat_host[m][n];
      }
   }

   printf("Graph Build \n");

   // ----------------------------------------------------
   // Decoder
   // ----------------------------------------------------
   int * CtoV_host, * VtoC_host, * Codeword, * Receivedword_host, * Decide_host, * U, l, kk, * CtoV_device, * VtoC_device, * Receivedword_device, * Decide_device;
   int iter;
   int * Synd_host, * Synd_device, * Interleaver_device;
   int Synd_host1 = 0;
   Synd_host = (int * ) calloc(M, sizeof(int));
   int varr;
   
   if (rand() % 100 >= 80) {
      varr = 1;
   } 
   else {
      varr = 0;
   }

   //Allocating memory for variables on device as well as the host
   hipMalloc((void ** ) & Synd_device, M * sizeof(int));
   CtoV_host = (int * ) calloc(NbBranch, sizeof(int));
   hipMalloc((void ** ) & CtoV_device, NbBranch * sizeof(int));
   VtoC_host = (int * ) calloc(NbBranch, sizeof(int));
   hipMalloc((void ** ) & VtoC_device, NbBranch * sizeof(int));
   Codeword = (int * ) calloc(N, sizeof(int));
   Receivedword_host = (int * ) calloc(N, sizeof(int));
   hipMalloc((void ** ) & Receivedword_device, N * sizeof(int));
   Decide_host = (int * ) calloc(N, sizeof(int));
   hipMalloc((void ** ) & Decide_device, N * sizeof(int));
   hipMalloc((void ** ) & Interleaver_device, NbBranch * sizeof(int));
   U = (int * ) calloc(N, sizeof(int));
   srand48(time(0) + Graine * 31 + 113);

   //Initializing grid and block dimensions

   dim3 GridDim1((N - 1) / 1024 + 1, 1);
   dim3 BlockDim1(1024);
   dim3 GridDim2((M - 1) / 1024 + 1, 1);
   dim3 BlockDim2(1024);

   // ----------------------------------------------------
   // Gaussian Elimination for the Encoding Matrix (Full Representation)
   // ----------------------------------------------------
   int ** MatFull, ** MatG, * PermG;
   int rank;

   MatG = (int ** ) calloc(M, sizeof(int * ));
   

   for (m = 0; m < M; m++) {
      MatG[m] = (int * ) calloc(N, sizeof(int));
   }
   
   MatFull = (int ** ) calloc(M, sizeof(int * ));
   
   for (m = 0; m < M; m++) {
      MatFull[m] = (int * ) calloc(N, sizeof(int));
   }

   PermG = (int * ) calloc(N, sizeof(int));

   for (n = 0; n < N; n++) {
      PermG[n] = n;
   }

   for (m = 0; m < M; m++) {
      for (k = 0; k < RowDegree[m]; k++) {
         MatFull[m][Mat_host[m][k]] = 1;
      }
   }
   rank = GaussianElimination_MRB(PermG, MatG, MatFull, M, N);

   // Variables for Statistics
   int IsCodeword, nb;
   int NiterMoy, NiterMax;
   int Dmin;
   int NbTotalErrors, NbBitError;
   int NbUnDetectedErrors, NbError;

   strcpy(FileName, FileResult);
   f = fopen(FileName, "w");
   fprintf(f, "-------------------------Gallager B--------------------------------------------------\n");
   fprintf(f, "alpha\t\tNbEr(BER)\t\tNbFer(FER)\t\tNbtested\t\tIterAver(Itermax)\tNbUndec(Dmin)\n");

   printf("-------------------------Gallager B--------------------------------------------------\n");
   printf("alpha\t\t\tNbEr(BER)\t\tNbFer(FER)\t\tNbtested\t\tIterAver(Itermax)\t\tNbUndec(Dmin)\n");

   // 
   for (alpha = alpha_max; alpha >= alpha_min; alpha -= alpha_step) {

      NiterMoy = 0;
      NiterMax = 0;
      Dmin = 1e5;
      NbTotalErrors = 0;
      NbBitError = 0;
      NbUnDetectedErrors = 0;
      NbError = 0;

      // Copying contents from the host to the device
      hipMemcpy(Interleaver_device, Interleaver_host, NbBranch * sizeof(int), hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(Mat_device), Mat_host1, NbBranch * sizeof(int));
      hipMemcpy(CtoV_device, CtoV_host, NbBranch * sizeof(int), hipMemcpyHostToDevice);
      hipMemcpy(VtoC_device, VtoC_host, NbBranch * sizeof(int), hipMemcpyHostToDevice);

      // encoding
#ifdef PROFILE 
  gettimeofday(&start,NULL);
#endif 
      for (nb = 0, nbtestedframes = 0; nb < NbMonteCarlo; nb++) {
         
         //
         memset(U,0,rank*sizeof(int));
         
         // randomly gerenates a uniform distribution of 0s and 1s
         for (k = rank; k < N; k++) {
            U[k] = floor(drand48() * 2);
         }

         // TODO this is what takes ~60% of the whole program
         for (k = rank - 1; k >= 0; k--) {
            for (l = k + 1; l < N; l++) {
               U[k] = U[k] ^ (MatG[k][l] * U[l]);
            }
         }

         //
         for (k = 0; k < N; k++) {
            Codeword[PermG[k]] = U[k];
         }
         
         // All zero codeword
         //for (n=0;n<N;n++) { Codeword[n]=0; }

         // Add Noise 
         for (n = 0; n < N; n++){
            if (drand48() < alpha){
               Receivedword_host[n] = 1 - Codeword[n];
            } 
            else {
               Receivedword_host[n] = Codeword[n];
            }
         }

         //============================================================================
         // Decoder
         //============================================================================
         
         //
         memset(CtoV_host,0,NbBranch*sizeof(int));

         //
         memmove(Decide_host,Receivedword_host,N*sizeof(int));

         hipMemcpy(Receivedword_device, Receivedword_host, N * sizeof(int), hipMemcpyHostToDevice);
         hipMemcpy(Decide_device, Decide_host, N * sizeof(int), hipMemcpyHostToDevice);

         for (iter = 0; iter < NbIter; iter++) {
            
            // Different itterations have different kernels
            if (iter == 0) {
               DataPassGB_0<<<GridDim1, BlockDim1>>>(VtoC_device, Receivedword_device, Interleaver_device, N, NbBranch);
            }
            else if (iter < 15) {
               DataPassGB_1<<<GridDim1, BlockDim1>>>(VtoC_device, CtoV_device, Receivedword_device, Interleaver_device, N, NbBranch);
            }
            else {
               DataPassGB_2<<<GridDim1, BlockDim1>>>(VtoC_device, CtoV_device, Receivedword_device, Interleaver_device, N, NbBranch, varr);
            }

            CheckPassGB<<<GridDim2, BlockDim2>>>(CtoV_device, VtoC_device, M, NbBranch);

            APP_GB<<<GridDim1, BlockDim1>>>(Decide_device, CtoV_device, Receivedword_device, Interleaver_device, N, NbBranch);

            ComputeSyndrome<<<GridDim2, BlockDim2>>>(Synd_device, Decide_device, M, NbBranch);

            hipMemcpy(Synd_host, Synd_device, M * sizeof(int), hipMemcpyDeviceToHost);

            // 
            int count1 = 0;
            for (kk = 0; kk < M; kk++) {
               if (Synd_host[kk] == 1) {
                  count1++;
                  break;
               }
            }

            // 
            if (count1 > 0) {
               Synd_host1 = 0;
            }           
            else {
               Synd_host1 = 1;
            }

            // if (IsCodeword) algorithm has converged and we are done, exit the loop
            IsCodeword = Synd_host1;
            if (IsCodeword) {
               break;
            }
         }

         hipMemcpy(Decide_host, Decide_device, N * sizeof(int), hipMemcpyDeviceToHost);

         //============================================================================
         // Compute Statistics
         //============================================================================
         nbtestedframes++;
         NbError = 0;

         //
         for (k = 0; k < N; k++) {
            if (Decide_host[k] != Codeword[k]) {
               ++NbError;
            }
         }

         // 
         NbBitError = NbBitError + NbError;
         
         // Case Divergence
         if (!IsCodeword) {
            NiterMoy = NiterMoy + NbIter;
            NbTotalErrors++;
         }

         // Case Convergence to Right Codeword
         if ((IsCodeword) && (NbError == 0)) {
            NiterMax = max(NiterMax, iter + 1);
            NiterMoy = NiterMoy + (iter + 1);
         }

         // Case Convergence to Wrong Codeword
         if ((IsCodeword) && (NbError != 0)) {
            NiterMax = max(NiterMax, iter + 1);
            NiterMoy = NiterMoy + (iter + 1);
            NbTotalErrors++;
            NbUnDetectedErrors++;
            Dmin = min(Dmin, NbError);
         }

         // Stopping Criterion
         if (NbTotalErrors == NBframes) {
            break;
         }
      }

#ifdef PROFILE  
  gettimeofday(&stop,NULL);  
  diffTime = diff_time_usec(start,stop);  
  fprintf(stderr,"time for loops in MicroSec: %lu \n",diffTime);
#endif 

      printf("%1.5f\t\t", alpha);
      printf("%10d (%1.6f)\t\t", NbBitError, (float) NbBitError / N / nbtestedframes);
      printf("%4d (%1.6f)\t\t", NbTotalErrors, (float) NbTotalErrors / nbtestedframes);
      printf("%10d\t\t", nbtestedframes);
      printf("%1.2f(%d)\t\t", (float) NiterMoy / nbtestedframes, NiterMax);
      printf("%d(%d)\n", NbUnDetectedErrors, Dmin);

      fprintf(f, "%1.5f\t\t", alpha);
      fprintf(f, "%10d (%1.8f)\t\t", NbBitError, (float) NbBitError / N / nbtestedframes);
      fprintf(f, "%4d (%1.8f)\t\t", NbTotalErrors, (float) NbTotalErrors / nbtestedframes);
      fprintf(f, "%10d\t\t", nbtestedframes);
      fprintf(f, "%1.2f(%d)\t\t", (float) NiterMoy / nbtestedframes, NiterMax);
      fprintf(f, "%d(%d)\n", NbUnDetectedErrors, Dmin);
   }

   //Freeing memory on the GPU
   hipFree(CtoV_device);
   hipFree(VtoC_device);
   hipFree(Interleaver_device);
   hipFree(Synd_device);
   hipFree(Receivedword_device);
   hipFree(Decide_device);
   fclose(f);

   return 0;
}
