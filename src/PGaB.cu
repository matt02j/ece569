#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////
// Created By        : Adithya Madhava Rao, Harshil Pankaj Kakaiya, Priyanka Devashish Goswami
// Modified By       : Matt <LastName>, Jeremy Seers, Sebastian Thiem
//                   :
// Organization:     : The University of Arizona
//                   :
// Project Name:     : OPTIMIZATIONS OF LDPC DECODERS IN CUDA
// File Name:        : PGaB.cu
//                   :
// Create Date:      : 8 May 2017
// Modified          : 24 March 2019
//                   :
// Description:      : Probabalistic Gallager B
//                   :
///////////////////////////////////////////////////////////////////////////////////////////////////

#include <string>
#include <iostream>
#include <fstream>
#include <random>

#include "const.cuh"
#include "utils.cuh"
#include "kernels.cuh"

// TODO Verify stride paterns
// All of the following functions use this
// for (unsigned stride = 0; stride < (num_branches / N); stride++)
// pattern and this seems inherently a little weird that each
// thread is striding 1, 2, 3, 4... away from itself...
// 
// TODO The Data pass modules and their 2 for loops essentially 
// read CtoV[node_idx] across "strides" different elements, and 
// throws away data so the next for loop has to read global again. 
// Maybe find a way to keep these values in shared memory. This may 
// be weird because of the striding pattern, but if that can be 
// addressed this opens up for us
//
// TODO once we model what these functions are doing 
// COMMENT
// 
// TODO GaussianElimination_MRB as a kernel
//
// TODO Main is reading args in the most horendous way ive
// ever seen, us fstreams, arg parser, ect anything to make
// that not what it is
// also uses 4 different input files, 3 of which are only dimensions
//
// TODO replace calloc with mallocs where possible
//
// also uses 4 different input files, 3 of which are only dimensions
// TODO replace calloc with mallocs where possible
//
// TODO make sure all allocated memory gets freed at the apropriate time

int main(int argc, char * argv[]) {

   // need address of compressed data matrix
   if(argc > 1){

#ifdef PROFILE
      struct timeval start, stop;
      unsigned long diffTime = 0;
#endif

#ifdef QUIET
      std::ofstream outFile("results.log");
      std::cout.rdbuf(outFile.rdbuf());
#endif

#ifdef VERBOSE
      std::cout << "Starting." << std::endl << std::endl;
#endif

      // --------------------------------------------------------------------------
      // Parameters and memory
      // --------------------------------------------------------------------------

      //-------------------------Reading command line arguments-------------------------
      std::string matrixAddr(argv[1]);    // convert data addr to a string

      //-------------------------Simulation parameters for PGaB-------------------------
      unsigned NbMonteCarlo = 1000000; // Maximum number of codewords sent
      unsigned itteration_count = 100; // Maximum nb of iterations
      unsigned frames_tested = 0;      // NOTE dont move this
      unsigned frame_count = 100;      // Simulation stops when frame_count in error

      //-------------------------Channels probability for bit-flip-------------------------
      float alpha = 0.01;        // NOTE leave this here...
      float alpha_max = 0.03;    // max alpha val
      float alpha_min = 0.03;    // min alpha value
      float alpha_step = 0.01;   // step size in alpha for loop
      
      //--------------------------------Random Number Generation--------------------------------
#ifdef TRUERANDOM
      std::random_device rd;                          // boilerplate
      std::mt19937 e2(rd());                          // boilerplate 
#else
      unsigned seed = 1337;                           // Seed Initialization for consistent Simulations
      std::mt19937 e2(seed);                          // boilerplate

#endif
      std::uniform_real_distribution<> dist(0, 1);    // uni{} (use dist(e2) to generate)

      //-------------------------Host memory structures-------------------------
      unsigned* h_matrix_flat;      // unrolled matrix
      unsigned* h_interleaver;      // ...
      unsigned char* h_CtoV;                  // ...
      unsigned char* h_VtoC;                  // ...
      unsigned char* h_messageRecieved;  // message after bit-flipping noise is applied
      unsigned char* h_decoded;          // message decoded
      unsigned char* h_synd;                  // ...
      unsigned char* h_bit_stream;       // Randomly generated bit stream 
      unsigned** h_MatG;            // ...
      unsigned char* h_MatG_flat;        // MatG flattened    
      unsigned bin_size = 0;

	#ifdef NUMSTREAMS
		hipStream_t s1,s2;
		hipEvent_t start1, stop1, start2, stop2;
		unsigned char* h_CtoV1;                  // ...
      		unsigned char* h_VtoC1;                  // ...
      		unsigned char* h_messageRecieved1;  // message after bit-flipping noise is applied
      		unsigned char* h_decoded1;          // message decoded
      		unsigned char* h_synd1;                  // ...
      		unsigned char* h_bit_stream1;       // Randomly generated bit stream 
      		unsigned char* d_CtoV1;                  // 
      		unsigned char* d_VtoC1;                  // 
      		unsigned char* d_messageRecieved1;  // message after bit-flipping noise is applied
      		unsigned char* d_decoded1;          // message decoded
      		unsigned char* d_synd1;                  // ...
      		unsigned char* d_bit_stream1;       // Randomly generated bit stream
      		unsigned* message1;         // test message {0,1,0,0,1,0,...} 
	#endif

      //-------------------------Device memory structures-------------------------
      // unsigned* d_matrix_flat;      // held as global in constant memory
      unsigned* d_interleaver;      // ...
      unsigned char* d_CtoV;                  // 
      unsigned char* d_VtoC;                  // 
      unsigned char* d_messageRecieved;  // message after bit-flipping noise is applied
      unsigned char* d_decoded;          // message decoded
      unsigned char* d_synd;                  // ...
      unsigned char* d_bit_stream;       // Randomly generated bit stream 
      unsigned char* d_MatG;             // MatG flattened
      unsigned int *d_Bins;

      //-------------------------Intermediate data structures-------------------------
      unsigned* rowRanks;        // list of codewords widths
      unsigned** data_matrix;    // matrix of codewords on the host
      unsigned* hist;            // histogram for <unk> // pinned
      unsigned* message;         // test message {0,1,0,0,1,0,...}
      unsigned** sparse_matrix;  // uncompressed sparse data matrix
      unsigned* PermG;           // array to keep track of permutations in h_MatG 

      //-------------------------Block and Grid dimensionality structures-------------------------
      dim3 GridDim1((N - 1) / BLOCK_DIM_1 + 1, 1);
      dim3 BlockDim1(BLOCK_DIM_1);
      dim3 GridDim2((M - 1) / BLOCK_DIM_2 + 1, 1);
      dim3 BlockDim2(BLOCK_DIM_2);
      dim3 NestedBlock(1024);
      dim3 NestedGrid(1);

#ifdef VERBOSE
      std::cout << "Reading in test data...";
#endif

      // Basically M*ColWidth but this code allows 
      // for their to be staggered columns, 
      // so the calculation is not as simple
      unsigned num_branches;

      // allocate and get row ranks
      rowRanks = (unsigned*)malloc(M * sizeof(unsigned));
      readRowRanks(rowRanks, M, (matrixAddr + "_RowDegree").c_str());

      // alocate and read in test data matrix from local file (also get num_branches while were in this loop)
      unsigned cols = 0;
      data_matrix = (unsigned**)malloc(M * sizeof(unsigned*));
      for (unsigned m = 0; m < M; m++) {
         cols = rowRanks[m];
         num_branches += cols;
         data_matrix[m] = (unsigned*)malloc(cols * sizeof(unsigned));
      }
      readDataMatrix(data_matrix, rowRanks, M, matrixAddr.c_str());

#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Allocating memory...";
#endif
#ifdef PROFILE 
      gettimeofday(&start, NULL); 
#endif
      //-------------------------Host Allocations-------------------------
      h_matrix_flat = (unsigned*)malloc(num_branches * sizeof(unsigned));
      h_interleaver = (unsigned*)malloc(num_branches * sizeof(unsigned));
      //h_synd = (unsigned char*)calloc(M, sizeof(unsigned char));
      //h_CtoV = (unsigned char*)calloc(num_branches, sizeof(unsigned char));
      //h_VtoC = (unsigned char*)calloc(num_branches, sizeof(unsigned char));
      //h_messageRecieved = (unsigned char*)calloc(N, sizeof(unsigned char));
      //h_decoded = (unsigned char*)calloc(N, sizeof(unsigned char));
      //h_bit_stream = (unsigned char *)calloc(N, sizeof(unsigned char));
      h_MatG = (unsigned **)calloc(M, sizeof(unsigned *));
      for (unsigned m = 0; m < M; m++) {
         h_MatG[m] = (unsigned *)calloc(N, sizeof(unsigned));
      }
      h_MatG_flat = (unsigned char*)malloc(M*N * sizeof(unsigned char));
	hipHostMalloc((void**)&h_synd,M*sizeof(unsigned char));
	hipHostMalloc((void**)&h_bit_stream,N*sizeof(unsigned char));
	
	hipHostMalloc((void**)&h_CtoV,num_branches*sizeof(unsigned char));
	hipHostMalloc((void**)&h_VtoC,num_branches*sizeof(unsigned char));
	hipHostMalloc((void**)&h_messageRecieved,N*sizeof(unsigned char));
	hipHostMalloc((void**)&h_decoded,N*sizeof(unsigned char));
      //-------------------------Device Allocations-------------------------
      hipMalloc((void**)&d_interleaver, num_branches * sizeof(unsigned));
      hipMalloc((void**)&d_synd, M * sizeof(unsigned char));
      hipMalloc((void**)&d_CtoV, num_branches * sizeof(unsigned char));
      hipMalloc((void**)&d_VtoC, num_branches * sizeof(unsigned char));
      hipMalloc((void**)&d_messageRecieved, N * sizeof(unsigned char));
      hipMalloc((void**)&d_decoded, N * sizeof(unsigned char));
      hipMalloc((void **)&d_bit_stream, N * sizeof(unsigned char));
      hipMalloc((void **)&d_MatG, M * N * sizeof(unsigned char));
      bin_size = sizeof(unsigned) * N;
      hipHostMalloc((void**)&hist, bin_size); // host pinned
      hipMalloc((void**) &d_Bins, bin_size);

      //-------------------------Other Allocations-------------------------
	#ifdef NUMSTREAMS
	hipEventCreate( &start1);
	hipEventCreate( &start2);
	hipStreamCreate( &s1 );
	hipStreamCreate( &s2 );
	hipHostMalloc((void**)&h_CtoV1,num_branches*sizeof(unsigned char));
	hipHostMalloc((void**)&h_VtoC1,num_branches*sizeof(unsigned char));
	hipHostMalloc((void**)&h_messageRecieved1,N*sizeof(unsigned char));
	hipHostMalloc((void**)&h_decoded1,N*sizeof(unsigned char));
	hipHostMalloc((void**)&h_synd1,M*sizeof(unsigned char));
	hipHostMalloc((void**)&h_bit_stream1,N*sizeof(unsigned char));
     	hipMalloc((void**)&d_synd1, M * sizeof(unsigned char));
      	hipMalloc((void**)&d_CtoV1, num_branches * sizeof(unsigned char));
     	hipMalloc((void**)&d_VtoC1, num_branches * sizeof(unsigned char));
     	hipMalloc((void**)&d_messageRecieved1, N * sizeof(unsigned char));
      	hipMalloc((void**)&d_decoded1, N * sizeof(unsigned char));
      	hipMalloc((void **)&d_bit_stream1, N * sizeof(unsigned char));
        message1 = (unsigned *)calloc(N, sizeof(unsigned));

	#endif
      message = (unsigned *)calloc(N, sizeof(unsigned));

      sparse_matrix = (unsigned **)calloc(M, sizeof(unsigned *));
      for (unsigned m = 0; m < M; m++) {
         sparse_matrix[m] = (unsigned *)calloc(N, sizeof(unsigned));
      }

      PermG = (unsigned *)calloc(N, sizeof(unsigned));


#ifdef VERBOSE
      std::cout << "Done." << std::endl; 

      std::cout << "Performing preliminary calulations...";
#endif

      // unroll host matrix into a flat host vector
      unrollMatrix(h_matrix_flat, data_matrix, rowRanks, M, num_branches);

      // Copying contents from the host to the device

      hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_flat), h_matrix_flat, num_branches * sizeof(unsigned));
 
      // generate histogram on the data matrix

      histogram_private_kernel<<<GridDim1, BlockDim1,N * sizeof(unsigned int)>>>(d_Bins, num_branches, N);
      hipMemcpy(hist, d_Bins, bin_size, hipMemcpyDeviceToHost);

      // generate interleaver
      initInterleaved(h_interleaver, data_matrix, rowRanks, hist, M, N);
      hipMemcpy(d_interleaver, h_interleaver, num_branches * sizeof(unsigned), hipMemcpyHostToDevice);


      // free no longer needed structures
      hipFree(hist);

      // init permutation matrix
      for (unsigned n = 0; n < N; n++) {
         PermG[n] = n;
      }

      // convert compressed data matrix to sparse matrix
      for (unsigned m = 0; m < M; m++) {
         for (unsigned k = 0; k < rowRanks[m]; k++) {
            sparse_matrix[m][data_matrix[m][k]] = 1;
         }
      }

#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Running Gaussian Elimination...";
#endif


      unsigned rank;

      rank = GaussianElimination_MRB(PermG, h_MatG, sparse_matrix, M, N);

      // free no longer needed data structures
      free2d(sparse_matrix, M);
      free2d(data_matrix, M);
      free(rowRanks);

#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Running Sim." << std::endl << std::endl;
#endif

      std::cout << "-------------------------------------------Gallager B-------------------------------------------" << std::endl;
      std::cout << "alpha\tNbEr(BER)\tNbFer(FER)\tNbtested\tIterAver(Itermax)\tNbUndec(Dmin)" << std::endl;

      // Variables for monitoring statistics
      unsigned err_total_count;
      unsigned bit_error_count;
      unsigned missed_error_count;
      unsigned err_count;
      unsigned NiterMoy;
      unsigned NiterMax;
      unsigned Dmin;

      // add stochastic element to itteratcions past 16
      unsigned varr = (dist(e2) <= 20) ? 1 : 0;

      // Flatten for memcpy // if we edit the gausian elimination function we can get rid of this
      for (unsigned m = 0; m < M; m++) {
         for (unsigned n = 0; n < N; n++) {
            h_MatG_flat[m * N + n] = (unsigned char)h_MatG[m][n];
         }
      } 


      // copy h_MatG_flat to device only once
      hipMemcpyAsync(d_MatG, h_MatG_flat, M * N * sizeof(unsigned char), hipMemcpyHostToDevice);

      // loop from alpha max to alpha min (increasing noise)
      for (alpha = alpha_max; alpha >= alpha_min; alpha -= alpha_step) {

         NiterMoy = 0;
         NiterMax = 0;
         Dmin = 1e5;
         err_total_count = 0;
         bit_error_count = 0;
         missed_error_count = 0;
         err_count = 0;

         //these are both all 0s?  // change to cuda memset
         hipMemcpyAsync(d_CtoV, h_CtoV, num_branches * sizeof(unsigned char), hipMemcpyHostToDevice,s1);
         hipMemcpyAsync(d_VtoC, h_VtoC, num_branches * sizeof(unsigned char), hipMemcpyHostToDevice,s1);
	 #ifdef NUMSTREAMS
         hipMemcpyAsync(d_CtoV1, h_CtoV, num_branches * sizeof(unsigned char), hipMemcpyHostToDevice,s2);
         hipMemcpyAsync(d_VtoC1, h_VtoC, num_branches * sizeof(unsigned char), hipMemcpyHostToDevice,s2);
	 #endif
         frames_tested = 0;
         unsigned nb = 0;
         while (nb < NbMonteCarlo && err_total_count != frame_count) {

            //--------------------------------------------Encode--------------------------------------------
#ifdef ZERO_CODE
            // All zero codeword
            for (n = 0; n < N; n++) {
               message[n] = 0;
            }
#else
            //
            memset(h_bit_stream, 0, rank * sizeof(unsigned char));

            // randomly gerenates a uniform distribution of 0s and 1s
            for (unsigned k = rank; k < N; k++) {
               h_bit_stream[k] = (unsigned char)floor(dist(e2) * 2);
            }

            //replace that super long loop
            hipMemcpyAsync(d_bit_stream, h_bit_stream, N * sizeof(unsigned char), hipMemcpyHostToDevice,s1);
            NestedFor <<<NestedGrid, NestedBlock, N * sizeof(unsigned char),s1 >>>(d_MatG, d_bit_stream, rank - 1, N);
            hipMemcpyAsync(h_bit_stream, d_bit_stream, N * sizeof(unsigned char), hipMemcpyDeviceToHost,s1);
            
           

	#ifdef NUMSTREAM
            //
            memset(h_bit_stream1, 0, rank * sizeof(unsigned char));

            // randomly gerenates a uniform distribution of 0s and 1s
            for (unsigned k = rank; k < N; k++) {
               h_bit_stream1[k] = (unsigned char)floor(dist(e2) * 2);
            }
            hipMemcpyAsync(d_bit_stream1, h_bit_stream1, N * sizeof(unsigned char), hipMemcpyHostToDevice,s2);
            NestedFor <<<NestedGrid, NestedBlock, N * sizeof(unsigned char),s2 >>>(d_MatG, d_bit_stream1, rank - 1, N);
            hipMemcpyAsync(h_bit_stream1, d_bit_stream1, N * sizeof(unsigned char), hipMemcpyDeviceToHost,s2);
            
	#endif 
		hipStreamSynchronize(s1);
	    for (unsigned k = 0; k < N; k++) {
               message[PermG[k]] = h_bit_stream[k];
            }
		
#endif
            //---------------------------------------Simulate Channel---------------------------------------

            // Flip the bits with the alpha percentage (noise over channel)
            for (unsigned n = 0; n < N; n++) {
               if (dist(e2) < alpha) {
                  h_messageRecieved[n] = 1 - message[n];
               }
               else {
                  h_messageRecieved[n] = message[n];
               }
            }
		hipStreamSynchronize(s2);
            for (unsigned k = 0; k < N; k++) {
               message1[PermG[k]] = h_bit_stream1[k];
            }
	    for (unsigned n = 0; n < N; n++) {
               if (dist(e2) < alpha) {
                  h_messageRecieved1[n] = 1 - message1[n];
               }
               else {
                  h_messageRecieved1[n] = message1[n];
               }
            }
            //-----------------------------------------------Decode-----------------------------------------------
            
            //
            //memmove(h_decoded, h_messageRecieved, N * sizeof(unsigned));

            hipMemcpyAsync(d_messageRecieved, h_messageRecieved, N * sizeof(unsigned char), hipMemcpyHostToDevice,s1);
            hipMemcpyAsync(d_messageRecieved1, h_messageRecieved1, N * sizeof(unsigned char), hipMemcpyHostToDevice,s2);
            //hipMemcpy(d_decoded, h_decoded, N * sizeof(unsigned), hipMemcpyHostToDevice);
            unsigned itter = 0;
            bool hasConverged = false;
            bool hasConverged1 = false;
            while (itter < itteration_count && (!hasConverged || !hasConverged1)) {

               // Different itterations have different kernels
               if (itter == 0) {
                  if(!hasConverged)DataPassGB_0 << <GridDim1, BlockDim1 ,0,s1>> > (d_VtoC, d_messageRecieved, d_interleaver, N, num_branches);
                  if(!hasConverged1)DataPassGB_0 << <GridDim1, BlockDim1 ,0,s2>> > (d_VtoC, d_messageRecieved, d_interleaver, N, num_branches);
               }
               else if (itter < 15) {
                  if(!hasConverged)DataPassGB_1 << <GridDim1, BlockDim1, num_branches*sizeof(unsigned char) + num_branches*sizeof(unsigned) ,s1>> > (d_VtoC, d_CtoV, d_messageRecieved, d_interleaver, N, num_branches);
                 if(!hasConverged1) DataPassGB_1 << <GridDim1, BlockDim1, num_branches*sizeof(unsigned char) + num_branches*sizeof(unsigned) ,s2>> > (d_VtoC1, d_CtoV1, d_messageRecieved1, d_interleaver, N, num_branches);
               }
               else {
                  if(!hasConverged)DataPassGB_2 << <GridDim1, BlockDim1 ,0,s1>> > (d_VtoC, d_CtoV, d_messageRecieved, d_interleaver, N, num_branches, varr);
                  if(!hasConverged1)DataPassGB_2 << <GridDim1, BlockDim1 ,0,s2>> > (d_VtoC1, d_CtoV1, d_messageRecieved1, d_interleaver, N, num_branches, varr);
               }

               if(!hasConverged)CheckPassGB << <GridDim2, BlockDim2, num_branches * sizeof(unsigned char),s1 >> > (d_CtoV, d_VtoC, M, num_branches);
               if(!hasConverged1)CheckPassGB << <GridDim2, BlockDim2, num_branches * sizeof(unsigned char),s2 >> > (d_CtoV1, d_VtoC1, M, num_branches);

               if(!hasConverged)APP_GB << <GridDim1, BlockDim1 ,0,s1>> > (d_decoded,d_CtoV, d_messageRecieved, d_interleaver, N, num_branches);
               if(!hasConverged1)APP_GB << <GridDim1, BlockDim1 ,0,s2>> > (d_decoded1,d_CtoV1, d_messageRecieved1, d_interleaver, N, num_branches);

              if(!hasConverged) ComputeSyndrome << <GridDim2, BlockDim2, N * sizeof(unsigned char),s1 >> > (d_synd, d_decoded, M, num_branches, N);
               if(!hasConverged)hipMemcpyAsync(h_synd, d_synd, M * sizeof(unsigned char), hipMemcpyDeviceToHost,s1);
               if(!hasConverged1)ComputeSyndrome << <GridDim2, BlockDim2, N * sizeof(unsigned char),s2 >> > (d_synd1, d_decoded1, M, num_branches, N);

               if(!hasConverged1)hipMemcpyAsync(h_synd1, d_synd1, M * sizeof(unsigned char), hipMemcpyDeviceToHost,s2);
 
		if(!hasConverged){
		       // check for convergence
			hipStreamSynchronize(s1);
		       hasConverged = true;
		       for (unsigned kk = 0; kk < M; kk++) {
		          if (h_synd[kk] == 1) {
		             hasConverged = false;
		             break;
		          }
		       }
		}
		if(!hasConverged1){
			hipStreamSynchronize(s2);
		       hasConverged1 = true;
		       for (unsigned kk = 0; kk < M; kk++) {
		          if (h_synd[kk] == 1) {
		             hasConverged1 = false;
		             break;
		          }
		       }
		}
               itter++;
            }

            hipMemcpyAsync(h_decoded, d_decoded, N * sizeof(unsigned char), hipMemcpyDeviceToHost,s1);
            hipMemcpyAsync(h_decoded1, d_decoded1, N * sizeof(unsigned char), hipMemcpyDeviceToHost,s2);

            //============================================================================
            // Compute Statistics
            //============================================================================
            frames_tested+=2;
            err_count = 0;

		hipStreamSynchronize(s1);
            // Calculate bit errors
            for (unsigned k = 0; k < N; k++) {
               if (h_decoded[k] != message[k]) { 
                  err_count++;
               }
            }
            bit_error_count += err_count;

            // Case Divergence
            if (!hasConverged) {
               NiterMoy = NiterMoy + itteration_count;
               err_total_count++;
            }
            // Case Convergence to Right message
            else if (err_count == 0) {
               NiterMax = max(NiterMax, itter + 1);
               NiterMoy = NiterMoy + (itter + 1);
            }
           // Case Convergence to Wrong message
            else{
               NiterMax = max(NiterMax, itter + 1);
               NiterMoy = NiterMoy + (itter + 1);
               err_total_count++;
               missed_error_count++;
               Dmin = min(Dmin, err_count);
            }

		hipStreamSynchronize(s2);
		err_count=0;
           // Calculate bit errors
            for (unsigned k = 0; k < N; k++) {
               if (h_decoded1[k] != message1[k]) { 
                  err_count++;
               }
            }
            bit_error_count += err_count;

            // Case Divergence
            if (!hasConverged) {
               NiterMoy = NiterMoy + itteration_count;
               err_total_count++;
            }
            // Case Convergence to Right message
            else if (err_count == 0) {
               NiterMax = max(NiterMax, itter + 1);
               NiterMoy = NiterMoy + (itter + 1);
            }
           // Case Convergence to Wrong message
            else{
               NiterMax = max(NiterMax, itter + 1);
               NiterMoy = NiterMoy + (itter + 1);
               err_total_count++;
               missed_error_count++;
               Dmin = min(Dmin, err_count);
            }

            nb++;
         }

#ifdef PROFILE  
         gettimeofday(&stop, NULL);
         diffTime = diff_time_usec(start, stop);
         fprintf(stderr, "time %lu \n", diffTime);
#endif 

         std::cout << alpha << "\t";
         std::cout << bit_error_count << "(" << (float)bit_error_count / N / frames_tested << ")  ";
         std::cout << err_total_count << "(" << (float)err_total_count / frames_tested << ")\t";
         std::cout << frames_tested << "\t\t";
         std::cout << (float)NiterMoy / frames_tested << "(" << NiterMax << ")\t\t";
         std::cout << missed_error_count << "(" << Dmin << ")\t" << std::endl;

      }

      //Freeing memory on the GPU
      hipFree(d_CtoV);
      hipFree(d_VtoC);
      hipFree(d_interleaver);
      hipFree(d_synd);
      hipFree(d_messageRecieved);
      hipFree(d_decoded1);
      hipFree(d_CtoV1);
      hipFree(d_VtoC1);
      hipFree(d_synd1);
      hipFree(d_messageRecieved1);
      hipFree(d_decoded);
      hipFree(d_Bins);
	hipHostFree(h_bit_stream);
	hipHostFree(h_synd);
	hipHostFree(h_CtoV);
	hipHostFree(h_VtoC);
	hipHostFree(h_decoded);
	hipHostFree(h_messageRecieved);
	hipHostFree(h_bit_stream1);
	hipHostFree(h_synd1);
	hipHostFree(h_CtoV1);
	hipHostFree(h_VtoC1);
	hipHostFree(h_decoded1);
	hipHostFree(h_messageRecieved1);
   }
   else {
      fprintf(stderr, "Usage: PGaB /Path/To/Data/File");
   }

   return 0;
}
