#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////
// Created By        : Adithya Madhava Rao, Harshil Pankaj Kakaiya, Priyanka Devashish Goswami
// Modified By       : Matt <LastName>, Jeremy Seers, Sebastian Thiem
//                   :
// Organization:     : The University of Arizona
//                   :
// Project Name:     : OPTIMIZATIONS OF LDPC DECODERS IN CUDA
// File Name:        : PGaB.cu
//                   :
// Create Date:      : 8 May 2017
// Modified          : 24 March 2019
//                   :
// Description:      : Probabalistic Gallager B
//                   :
///////////////////////////////////////////////////////////////////////////////////////////////////

#include <string>
#include <iostream>
#include <fstream>
#include <random>

#include "const.cuh"
#include "utils.cuh"
#include "kernels.cuh"

// TODO Verify stride paterns
// All of the following functions use this
// for (unsigned stride = 0; stride < (num_branches / N); stride++)
// pattern and this seems inherently a little weird that each
// thread is striding 1, 2, 3, 4... away from itself...
// 
// TODO The Data pass modules and their 2 for loops essentially 
// read CtoV[node_idx] across "strides" different elements, and 
// throws away data so the next for loop has to read global again. 
// Maybe find a way to keep these values in shared memory. This may 
// be weird because of the striding pattern, but if that can be 
// addressed this opens up for us
//
// TODO once we model what these functions are doing 
// COMMENT
// 
// TODO GaussianElimination_MRB as a kernel
//
// TODO Main is reading args in the most horendous way ive
// ever seen, us fstreams, arg parser, ect anything to make
// that not what it is
// also uses 4 different input files, 3 of which are only dimensions
//
// TODO replace calloc with mallocs where possible
//
// also uses 4 different input files, 3 of which are only dimensions
// TODO replace calloc with mallocs where possible
//
// TODO make sure all allocated memory gets freed at the apropriate time

int main(int argc, char * argv[]) {

   // need address of compressed data matrix
   if(argc > 1){

#ifdef PROFILE
      struct timeval start, stop;
      unsigned long diffTime = 0;
#endif

#ifdef QUIET
      std::ofstream outFile("results.log");
      std::cout.rdbuf(outFile.rdbuf());
#endif

#ifdef VERBOSE
      std::cout << "Starting." << std::endl << std::endl;
#endif

      // --------------------------------------------------------------------------
      // Parameters and memory
      // --------------------------------------------------------------------------

      //-------------------------Reading command line arguments-------------------------
      std::string matrixAddr(argv[1]);    // convert data addr to a string

      //-------------------------Simulation parameters for PGaB-------------------------
      unsigned NbMonteCarlo = 1000000; // Maximum number of codewords sent
      unsigned itteration_count = 100; // Maximum nb of iterations
      unsigned frames_tested = 0;      // NOTE dont move this
      unsigned frame_count = 100;      // Simulation stops when frame_count in error

      //-------------------------Channels probability for bit-flip-------------------------
      float alpha = 0.01;        // NOTE leave this here...
      float alpha_max = 0.03;    // max alpha val
      float alpha_min = 0.03;    // min alpha value
      float alpha_step = 0.01;   // step size in alpha for loop
      
      //--------------------------------Random Number Generation--------------------------------
#ifdef TRUERANDOM
      std::random_device rd;                          // boilerplate
      std::mt19937 e2(rd());                          // boilerplate 
#else
      unsigned seed = 1337;                           // Seed Initialization for consistent Simulations
      std::mt19937 e2(seed);                          // boilerplate

#endif
      std::uniform_real_distribution<> dist(0, 1);    // uni{} (use dist(e2) to generate)

      //-------------------------Host memory structures-------------------------
      unsigned* h_matrix_flat;      // unrolled matrix
      unsigned* h_interleaver;      // ...
      int* h_CtoV;                  // ...
      int* h_VtoC;                  // ...
      unsigned* h_messageRecieved;  // message after bit-flipping noise is applied
      unsigned* h_decoded;          // message decoded
      int* h_synd;                  // ...
      unsigned* h_bit_stream;       // Randomly generated bit stream 
      unsigned** h_MatG;            // ...
      unsigned* h_MatG_flat;        // MatG flattened

      //-------------------------Device memory structures-------------------------
      // unsigned* d_matrix_flat;      // held as global in constant memory
      unsigned* d_interleaver;      // ...
      int* d_CtoV;                  // 
      int* d_VtoC;                  // 
      unsigned* d_messageRecieved;  // message after bit-flipping noise is applied
      unsigned* d_decoded;          // message decoded
      int* d_synd;                  // ...
      unsigned* d_bit_stream;       // Randomly generated bit stream 
      unsigned* d_MatG;             // MatG flattened

      //-------------------------Intermediate data structures-------------------------
      unsigned* rowRanks;        // list of codewords widths
      unsigned** data_matrix;    // matrix of codewords on the host
      unsigned* hist;            // histogram for <unk>
      unsigned* message;         // test message {0,1,0,0,1,0,...}
      unsigned** sparse_matrix;  // uncompressed sparse data matrix
      unsigned* PermG;           // array to keep track of permutations in h_MatG 

      //-------------------------Block and Grid dimensionality structures-------------------------
      dim3 GridDim1((N - 1) / BLOCK_DIM_1 + 1, 1);
      dim3 BlockDim1(BLOCK_DIM_1);
      dim3 GridDim2((M - 1) / BLOCK_DIM_2 + 1, 1);
      dim3 BlockDim2(BLOCK_DIM_2);
      dim3 NestedBlock(1024);
      dim3 NestedGrid(1);

#ifdef VERBOSE
      std::cout << "Reading in test data...";
#endif

      // Basically M*ColWidth but this code allows 
      // for their to be staggered columns, 
      // so the calculation is not as simple
      unsigned num_branches;

      // allocate and get row ranks
      rowRanks = (unsigned*)malloc(M * sizeof(unsigned));
      readRowRanks(rowRanks, M, (matrixAddr + "_RowDegree").c_str());

      // alocate and read in test data matrix from local file (also get num_branches while were in this loop)
      unsigned cols = 0;
      data_matrix = (unsigned**)malloc(M * sizeof(unsigned*));
      for (unsigned m = 0; m < M; m++) {
         cols = rowRanks[m];
         num_branches += cols;
         data_matrix[m] = (unsigned*)malloc(cols * sizeof(unsigned));
      }
      readDataMatrix(data_matrix, rowRanks, M, matrixAddr.c_str());

#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Allocating memory...";
#endif
      //-------------------------Host Allocations-------------------------
      h_matrix_flat = (unsigned*)malloc(num_branches * sizeof(unsigned));
      h_interleaver = (unsigned*)malloc(num_branches * sizeof(unsigned));
      h_synd = (int*)calloc(M, sizeof(int));
      h_CtoV = (int*)calloc(num_branches, sizeof(int));
      h_VtoC = (int*)calloc(num_branches, sizeof(int));
      h_messageRecieved = (unsigned*)calloc(N, sizeof(unsigned));
      h_decoded = (unsigned*)calloc(N, sizeof(unsigned));
      h_bit_stream = (unsigned *)calloc(N, sizeof(unsigned));
      h_MatG = (unsigned **)calloc(M, sizeof(unsigned *));
      for (unsigned m = 0; m < M; m++) {
         h_MatG[m] = (unsigned *)calloc(N, sizeof(unsigned));
      }
      h_MatG_flat = (unsigned*)malloc(M*N * sizeof(unsigned));

      //-------------------------Device Allocations-------------------------
      hipMalloc((void**)&d_interleaver, num_branches * sizeof(unsigned));
      hipMalloc((void**)&d_synd, M * sizeof(int));
      hipMalloc((void**)&d_CtoV, num_branches * sizeof(int));
      hipMalloc((void**)&d_VtoC, num_branches * sizeof(int));
      hipMalloc((void**)&d_messageRecieved, N * sizeof(unsigned));
      hipMalloc((void**)&d_decoded, N * sizeof(unsigned));
      hipMalloc((void **)&d_bit_stream, N * sizeof(unsigned));
      hipMalloc((void **)&d_MatG, M * N * sizeof(unsigned));

      //-------------------------Other Allocations-------------------------
      hist = (unsigned*)calloc(N, sizeof(unsigned));
      message = (unsigned *)calloc(N, sizeof(unsigned));

      sparse_matrix = (unsigned **)calloc(M, sizeof(unsigned *));
      for (unsigned m = 0; m < M; m++) {
         sparse_matrix[m] = (unsigned *)calloc(N, sizeof(unsigned));
      }

      PermG = (unsigned *)calloc(N, sizeof(unsigned));


#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Performing preliminary calulations...";
#endif

      // generate histogram on the data matrix
      histogram(hist, data_matrix, rowRanks, M, N);

      // generate interleaver
      initInterleaved(h_interleaver, data_matrix, rowRanks, hist, M, N);

      // unroll host matrix into a flat host vector
      unrollMatrix(h_matrix_flat, data_matrix, rowRanks, M, num_branches);

      // free no longer needed structures
      free(hist);

      // init permutation matrix
      for (unsigned n = 0; n < N; n++) {
         PermG[n] = n;
      }

      // convert compressed data matrix to sparse matrix
      for (unsigned m = 0; m < M; m++) {
         for (unsigned k = 0; k < rowRanks[m]; k++) {
            sparse_matrix[m][data_matrix[m][k]] = 1;
         }
      }

#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Running Gaussian Elimination...";
#endif
#ifdef PROFILE 
      gettimeofday(&start, NULL);
#endif

      unsigned rank;

      rank = GaussianElimination_MRB(PermG, h_MatG, sparse_matrix, M, N);

      // free no longer needed data structures
      free2d(sparse_matrix, M);
      free2d(data_matrix, M);
      free(rowRanks);

#ifdef VERBOSE
      std::cout << "Done." << std::endl;

      std::cout << "Running Sim." << std::endl << std::endl;
#endif

      std::cout << "-------------------------------------------Gallager B-------------------------------------------" << std::endl;
      std::cout << "alpha\tNbEr(BER)\tNbFer(FER)\tNbtested\tIterAver(Itermax)\tNbUndec(Dmin)" << std::endl;

      // Variables for monitoring statistics
      unsigned err_total_count;
      unsigned bit_error_count;
      unsigned missed_error_count;
      unsigned err_count;
      unsigned NiterMoy;
      unsigned NiterMax;
      unsigned Dmin;

      // add stochastic element to itteratcions past 16
      unsigned varr = (dist(e2) <= 20) ? 1 : 0;

      // Flatten for memcpy // if we edit the gausian elimination function we can get rid of this
      for (unsigned m = 0; m < M; m++) {
         for (unsigned n = 0; n < N; n++) {
            h_MatG_flat[m * N + n] = h_MatG[m][n];
         }
      }

      // copy h_MatG_flat to device only once
      hipMemcpyAsync(d_MatG, h_MatG_flat, M * N * sizeof(unsigned), hipMemcpyHostToDevice);

      // loop from alpha max to alpha min (increasing noise)
      for (alpha = alpha_max; alpha >= alpha_min; alpha -= alpha_step) {

         NiterMoy = 0;
         NiterMax = 0;
         Dmin = 1e5;
         err_total_count = 0;
         bit_error_count = 0;
         missed_error_count = 0;
         err_count = 0;

         // Copying contents from the host to the device
         hipMemcpy(d_interleaver, h_interleaver, num_branches * sizeof(unsigned), hipMemcpyHostToDevice);
         hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_flat), h_matrix_flat, num_branches * sizeof(unsigned));

         //these are both all 0s? 
         hipMemcpy(d_CtoV, h_CtoV, num_branches * sizeof(int), hipMemcpyHostToDevice);
         hipMemcpy(d_VtoC, h_VtoC, num_branches * sizeof(int), hipMemcpyHostToDevice);

         frames_tested = 0;
         unsigned nb = 0;
         while (nb < NbMonteCarlo && err_total_count != frame_count) {

            //--------------------------------------------Encode--------------------------------------------
#ifdef ZERO_CODE
            // All zero codeword
            for (n = 0; n < N; n++) {
               message[n] = 0;
            }
#else
            //
            memset(h_bit_stream, 0, rank * sizeof(unsigned));

            // randomly gerenates a uniform distribution of 0s and 1s
            for (unsigned k = rank; k < N; k++) {
               h_bit_stream[k] = floor(dist(e2) * 2);
            }

            //replace that super long loop
            hipMemcpy(d_bit_stream, h_bit_stream, N * sizeof(unsigned), hipMemcpyHostToDevice);
            NestedFor <<<NestedGrid, NestedBlock, N * sizeof(unsigned) >>>(d_MatG, d_bit_stream, rank - 1, N);
            hipMemcpy(h_bit_stream, d_bit_stream, N * sizeof(unsigned), hipMemcpyDeviceToHost);
            // TODO this is what takes ~60% of the whole program //obsolete
            //for (k = rank - 1; k >= 0; k--) {
            //   for (l = k + 1; l < N; l++) {
            //      h_bit_stream[k] = h_bit_stream[k] ^ (h_MatG[k][l] * h_bit_stream[l]);
            //   }
            //}

            //
            for (unsigned k = 0; k < N; k++) {
               message[PermG[k]] = h_bit_stream[k];
            }
#endif
            //---------------------------------------Simulate Channel---------------------------------------

            // Flip the bits with the alpha percentage (noise over channel)
            for (unsigned n = 0; n < N; n++) {
               if (dist(e2) < alpha) {
                  h_messageRecieved[n] = 1 - message[n];
               }
               else {
                  h_messageRecieved[n] = message[n];
               }
            }

            //-----------------------------------------------Decode-----------------------------------------------
            
            //
            memmove(h_decoded, h_messageRecieved, N * sizeof(unsigned));

            hipMemcpy(d_messageRecieved, h_messageRecieved, N * sizeof(unsigned), hipMemcpyHostToDevice);
            hipMemcpy(d_decoded, h_decoded, N * sizeof(unsigned), hipMemcpyHostToDevice);

            unsigned itter = 0;
            bool hasConverged = false;
            while (itter < itteration_count && !hasConverged) {

               // Different itterations have different kernels
               if (itter == 0) {
                  DataPassGB_0 << <GridDim1, BlockDim1 >> > (d_VtoC, d_messageRecieved, d_interleaver, N, num_branches);
               }
               else if (itter < 15) {
                  DataPassGB_1 << <GridDim1, BlockDim1 >> > (d_VtoC, d_CtoV, d_messageRecieved, d_interleaver, N, num_branches);
               }
               else {
                  DataPassGB_2 << <GridDim1, BlockDim1 >> > (d_VtoC, d_CtoV, d_messageRecieved, d_interleaver, N, num_branches, varr);
               }

               CheckPassGB << <GridDim2, BlockDim2, num_branches * sizeof(int) >> > (d_CtoV, d_VtoC, M, num_branches);

               APP_GB << <GridDim1, BlockDim1 >> > (d_decoded, d_CtoV, d_messageRecieved, d_interleaver, N, num_branches);

               ComputeSyndrome << <GridDim2, BlockDim2, N * sizeof(int) >> > (d_synd, d_decoded, M, num_branches, N);

               hipMemcpy(h_synd, d_synd, M * sizeof(int), hipMemcpyDeviceToHost);

               // 
               int count1 = 0;
               for (unsigned kk = 0; kk < M; kk++) {
                  if (h_synd[kk] == 1) {
                     count1++;
                     break;
                  }
               }

               // check for convergence
               hasConverged = true;
               for (unsigned kk = 0; kk < M; kk++) {
                  if (h_synd[kk] == 1) {
                     hasConverged = false;
                     break;
                  }
               }

               itter++;
            }

            hipMemcpy(h_decoded, d_decoded, N * sizeof(unsigned), hipMemcpyDeviceToHost);

            //============================================================================
            // Compute Statistics
            //============================================================================
            frames_tested++;
            err_count = 0;

            // Calculate bit errors
            for (unsigned k = 0; k < N; k++) {
               if (h_decoded[k] != message[k]) {
                  err_count++;
               }
            }
            bit_error_count += err_count;

            // Case Divergence
            if (!hasConverged) {
               NiterMoy = NiterMoy + itteration_count;
               err_total_count++;
            }

            // Case Convergence to Right message
            if ((hasConverged) && (err_count == 0)) {
               NiterMax = max(NiterMax, itter + 1);
               NiterMoy = NiterMoy + (itter + 1);
            }

            // Case Convergence to Wrong message
            if ((hasConverged) && (err_count != 0)) {
               NiterMax = max(NiterMax, itter + 1);
               NiterMoy = NiterMoy + (itter + 1);
               err_total_count++;
               missed_error_count++;
               Dmin = min(Dmin, err_count);
            }

            nb++;
         }

#ifdef PROFILE  
         gettimeofday(&stop, NULL);
         diffTime = diff_time_usec(start, stop);
         fprintf(stderr, " %lu \n", diffTime);
#endif 

         std::cout << alpha << "\t";
         std::cout << bit_error_count << "(" << (float)bit_error_count / N / frames_tested << ")  ";
         std::cout << err_total_count << "(" << (float)err_total_count / frames_tested << ")\t";
         std::cout << frames_tested << "\t\t";
         std::cout << (float)NiterMoy / frames_tested << "(" << NiterMax << ")\t\t";
         std::cout << missed_error_count << "(" << Dmin << ")\t" << std::endl;

      }

      //Freeing memory on the GPU
      hipFree(d_CtoV);
      hipFree(d_VtoC);
      hipFree(d_interleaver);
      hipFree(d_synd);
      hipFree(d_messageRecieved);
      hipFree(d_decoded);
   }
   else {
      fprintf(stderr, "Usage: PGaB /Path/To/Data/File");
   }

   return 0;
}
